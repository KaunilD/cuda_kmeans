#include "hip/hip_runtime.h"
#include "cuda_datum.hpp"

__global__ void d_cudaKMeans(float * datum){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	datum[id]+=datum[id];
}

void h_cudaKMeans(CUDADatum & datum){
	// 32 = warp size. shared memory is 
	// permitted between threads in a block
	int blockSize = 32*10; // number of threads in a block
	int gridSize = datum.size/blockSize; // number of blocks;
	d_cudaKMeans<<<dim3(gridSize), dim3(blockSize)>>>(datum.d_buffer);
}