#include "hip/hip_runtime.h"
#include "cuda_kmeans/cuda_datum.hpp"

__device__ float d_l2norm(float f1, float f2){
	return f1-f2;
}

__global__ void d_cudaKMeans(float * dataset_1, float * dataset_2, float * matches, int length){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id >= length){
		return;
	}
	float last_distance_matched = INT_MAX, curr_distance_matched = 0;
	
	matches[id] = 0.0;
	for(size_t i = 0; i < length; ++i){
		if (i == id){
			continue;
		}
		curr_distance_matched = d_l2norm(dataset_1[id], dataset_2[i]);
		if ( last_distance_matched > curr_distance_matched ){
			matches[id] = i;
			last_distance_matched = curr_distance_matched;
		}
	}

	return;
}

void h_cudaKMeans(CUDADatum * datum_1, CUDADatum * datum_2, CUDADatum * matches){
	// 32 = warp size. shared memory is 
	// permitted between threads in a block
	int blockSize = 32*10; // number of threads in a block
	int gridSize = (datum_1->length + blockSize - 1)/blockSize; // number of blocks;

	d_cudaKMeans<<<gridSize, blockSize>>>(
		datum_1->d_in_buffer, datum_2->d_in_buffer, matches->d_out_buffer, datum_1->length
	);

	hipDeviceSynchronize();
	
}